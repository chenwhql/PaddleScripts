#include "hip/hip_runtime.h"
// relu_op.cu
#include "paddle/fluid/framework/op_registry.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void KeRelu2(const T* x, const int num, T* y) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = gid; i < num; i += blockDim.x * gridDim.x) {
    y[i] = max(x[i], static_cast<T>(0.));
  }
}

// 前向OP的kernel的GPU实现
template <typename DeviceContext, typename T>
class Relu2CUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* in_t = ctx.Input<Tensor>("X");
    auto* out_t = ctx.Output<Tensor>("Y");
    auto x = in_t->data<T>();
    auto y = out_t->mutable_data<T>(ctx.GetPlace());

    auto& dev_ctx = ctx.template device_context<DeviceContext>();

    int num = in_t->numel();
    int block = 512;
    int grid = (num + block - 1) / block;
    KeRelu2<T><<<grid, block, 0, dev_ctx.stream()>>>(x, num, y);
  }
};

template <typename T>
__global__ void KeRelu2Grad(const T* y, const T* dy, const int num, T* dx) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = gid; i < num; i += blockDim.x * gridDim.x) {
    dx[i] = dy[i] * (y[i] > 0 ? 1. : 0.);
  }
}

// 反向OP的kernel的GPU实现
template <typename DeviceContext, typename T>
class Relu2GradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* dy_t = ctx.Input<Tensor>(framework::GradVarName("Y"));
    auto* y_t = ctx.Input<Tensor>("Y");
    auto* dx_t = ctx.Output<Tensor>(framework::GradVarName("X"));

    auto dy = dy_t->data<T>();
    auto y = y_t->data<T>();
    auto dx = dx_t->mutable_data<T>(ctx.GetPlace());

    auto& dev_ctx = ctx.template device_context<DeviceContext>();

    int num = dy_t->numel();
    int block = 512;
    int grid = (num + block - 1) / block;
    KeRelu2Grad<T><<<grid, block, 0, dev_ctx.stream()>>>(y, dy, num, dx);
  }
};

}  // namespace operators
}  // namespace paddle

using CUDA = paddle::platform::CUDADeviceContext;
// 注册前向的GPU Kernel
REGISTER_OP_CUDA_KERNEL(relu2,
                        paddle::operators::Relu2CUDAKernel<CUDA, float>,
                        paddle::operators::Relu2CUDAKernel<CUDA, double>);
// 注册反向的GPU Kernel
REGISTER_OP_CUDA_KERNEL(relu2_grad,
                        paddle::operators::Relu2GradCUDAKernel<CUDA, float>,
                        paddle::operators::Relu2GradCUDAKernel<CUDA, double>);
