#include "hip/hip_runtime.h"
#include "paddle/extension.h"

#include <vector>

#define CHECK_INPUT(x) PD_CHECK(x.place() == paddle::PlaceType::kGPU, #x " must be a GPU Tensor.")

template <typename data_t>
__global__ void relu_cuda_forward_kernel(const data_t* x,
                                         data_t* y,
                                         const int num) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = gid; i < num; i += blockDim.x * gridDim.x) {
    y[i] = max(x[i], static_cast<data_t>(0.));
  }
}

template <typename data_t>
__global__ void relu_cuda_backward_kernel(const data_t* dy,
                                          const data_t* y,
                                          data_t* dx,
                                          const int num) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = gid; i < num; i += blockDim.x * gridDim.x) {
    dx[i] = dy[i] * (y[i] > 0 ? 1. : 0.);
  }
}

std::vector<paddle::Tensor> relu_cuda_forward(const paddle::Tensor& x) {
  auto out = paddle::Tensor(paddle::PlaceType::kGPU, x.shape());

  int numel = x.size();
  int block = 512;
  int grid = (numel + block - 1) / block;
  PD_DISPATCH_FLOATING_TYPES(
      x.type(), "relu_cuda_forward_kernel", ([&] {
        relu_cuda_forward_kernel<data_t><<<grid, block, 0, x.stream()>>>(
            x.data<data_t>(), out.mutable_data<data_t>(x.place()), numel);
      }));

  return {out};
}

std::vector<paddle::Tensor> relu_cuda_backward(const paddle::Tensor& x,
                                               const paddle::Tensor& out,
                                               const paddle::Tensor& grad_out) {
  auto grad_x = paddle::Tensor(paddle::PlaceType::kGPU, x.shape());

  int numel = out.size();
  int block = 512;
  int grid = (numel + block - 1) / block;
  PD_DISPATCH_FLOATING_TYPES(
      out.type(), "relu_cuda_backward_kernel", ([&] {
        relu_cuda_backward_kernel<data_t><<<grid, block, 0, x.stream()>>>(
            grad_out.data<data_t>(),
            out.data<data_t>(),
            grad_x.mutable_data<data_t>(x.place()),
            numel);
      }));

  return {grad_x};
}

std::vector<paddle::Tensor> ReluCUDAForward(const paddle::Tensor& x) {
  CHECK_INPUT(x);

  return relu_cuda_forward(x);
}

std::vector<paddle::Tensor> ReluCUDABackward(const paddle::Tensor& x,
                                             const paddle::Tensor& out,
                                             const paddle::Tensor& grad_out) {
  CHECK_INPUT(x);
  CHECK_INPUT(out);
  CHECK_INPUT(grad_out);

  return relu_cuda_backward(x, out, grad_out);
}

PD_BUILD_OP(custom_relu)
    .Inputs({"X"})
    .Outputs({"Out"})
    .SetKernelFn(PD_KERNEL(ReluCUDAForward));

PD_BUILD_GRAD_OP(custom_relu)
    .Inputs({"X", "Out", paddle::Grad("Out")})
    .Outputs({paddle::Grad("X")})
    .SetKernelFn(PD_KERNEL(ReluCUDABackward));
